#include "hip/hip_runtime.h"
#include "CudaMatrixOperator.cuh"

__device__ __forceinline__
float getVectorNorm(float* v, int dim)
{
	return normf(dim, v);
}

__device__ __forceinline__
float getVectorDot(float* v1, float* v2, int dim)
{
	float sum = 0;
	for (uint32_t i = 0; i < dim; i++)
		sum += v1[i] * v2[i];
	return sum;
}

__device__ __forceinline__
void getVectorNormalize(float* v, int dim)
{
	float len = getVectorNorm(v, dim);
	for (uint32_t i = 0; i < dim; i++)
		v[i] /= len;
}

__device__ __forceinline__
void getVector3Cross(float* v1, float* v2, float* result)
{
	result[0] = v1[1] * v2[2] - v2[1] * v1[2];
	result[1] = v1[2] * v2[0] - v2[2] * v1[0];
	result[2] = v1[0] * v2[1] - v2[0] * v1[1];
}

__device__ __forceinline__
void getVectorSub(float* v1, float* v2, float* result, int dim)
{
	for (uint32_t i = 0; i < dim; i++)
		result[i] = v1[i] - v2[i];
}

__device__ __forceinline__
void getMutilMatrix(float* lmat, float* rmat, float* result, int dim, int rdim, int cdim)
{
	for (uint32_t c = 0; c < cdim; c++)
	{
		for (uint32_t r = 0; r < rdim; r++)
		{
			int idx = c * rdim + r;
			result[idx] = 0;
			for (uint32_t b = 0; b < dim; b++)
					result[idx] += lmat[b * rdim + r] * rmat[c * dim + b];
		}
	}
}

//un-safe & hard code
__device__ __forceinline__
float getMatrix3Determinant(float* mat)
{
	float determinant = mat[0] * mat[4] * mat[8] - mat[2] * mat[4] * mat[6] + mat[1] * mat[5] * mat[6] + mat[2] * mat[3] * mat[7] - mat[0] * mat[5] * mat[7] - mat[1] * mat[3] * mat[8];
	return determinant;
}

//un-safe & hard code
__device__ __forceinline__
void getMatrix3Inverse(float* mat, float* inv)
{
	float len = getMatrix3Determinant(mat);

	inv[0] = (mat[4] * mat[8] - mat[5] * mat[7]) / len; inv[3] = (mat[5] * mat[6] - mat[3] * mat[8]) / len; inv[6] = (mat[3] * mat[7] - mat[4] * mat[6]) / len;
	inv[1] = (mat[2] * mat[7] - mat[1] * mat[8]) / len; inv[4] = (mat[0] * mat[8] - mat[2] * mat[6]) / len; inv[7] = (mat[1] * mat[6] - mat[0] * mat[7]) / len;
	inv[2] = (mat[1] * mat[5] - mat[2] * mat[4]) / len; inv[5] = (mat[2] * mat[3] - mat[0] * mat[5]) / len; inv[8] = (mat[0] * mat[4] - mat[1] * mat[3]) / len;
}

//un-safe & hard code
__device__ __forceinline__
float getMatrix4Determinant(float* mat)
{
	float len = mat[1] * mat[11] * mat[14] * mat[4] - mat[1] * mat[10] * mat[15] * mat[4] -
		mat[11] * mat[13] * mat[2] * mat[4] + mat[10] * mat[13] * mat[3] * mat[4] -
		mat[0] * mat[11] * mat[14] * mat[5] + mat[0] * mat[10] * mat[15] * mat[5] +
		mat[11] * mat[12] * mat[2] * mat[5] - mat[10] * mat[12] * mat[3] * mat[5] -
		mat[1] * mat[11] * mat[12] * mat[6] + mat[0] * mat[11] * mat[13] * mat[6] +
		mat[1] * mat[10] * mat[12] * mat[7] - mat[0] * mat[10] * mat[13] * mat[7] -
		mat[15] * mat[2] * mat[5] * mat[8] + mat[14] * mat[3] * mat[5] * mat[8] + mat[1] * mat[15] * mat[6] * mat[8] -
		mat[13] * mat[3] * mat[6] * mat[8] - mat[1] * mat[14] * mat[7] * mat[8] + mat[13] * mat[2] * mat[7] * mat[8] +
		mat[15] * mat[2] * mat[4] * mat[9] - mat[14] * mat[3] * mat[4] * mat[9] - mat[0] * mat[15] * mat[6] * mat[9] +
		mat[12] * mat[3] * mat[6] * mat[9] + mat[0] * mat[14] * mat[7] * mat[9] - mat[12] * mat[2] * mat[7] * mat[9];
	return len;
}

//un-safe & hard code
__device__ __forceinline__
void getMatrix4Inverse(float* mat, float* inv)
{
	float len = getMatrix4Determinant(mat);
	inv[0] = (-mat[11] * mat[14] * mat[5] + mat[10] * mat[15] * mat[5] + mat[11] * mat[13] * mat[6] - mat[10] * mat[13] * mat[7] - mat[15] * mat[6] * mat[9] + mat[14] * mat[7] * mat[9]) / len;
	inv[1] = (mat[1] * mat[11] * mat[14] - mat[1] * mat[10] * mat[15] - mat[11] * mat[13] * mat[2] + mat[10] * mat[13] * mat[3] + mat[15] * mat[2] * mat[9] - mat[14] * mat[3] * mat[9]) / len;
	inv[2] = (-mat[15] * mat[2] * mat[5] + mat[14] * mat[3] * mat[5] + mat[1] * mat[15] * mat[6] - mat[13] * mat[3] * mat[6] - mat[1] * mat[14] * mat[7] + mat[13] * mat[2] * mat[7]) / len;
	inv[3] = (mat[11] * mat[2] * mat[5] - mat[10] * mat[3] * mat[5] - mat[1] * mat[11] * mat[6] + mat[1] * mat[10] * mat[7] + mat[3] * mat[6] * mat[9] - mat[2] * mat[7] * mat[9]) / len;
	inv[4] = (mat[11] * mat[14] * mat[4] - mat[10] * mat[15] * mat[4] - mat[11] * mat[12] * mat[6] + mat[10] * mat[12] * mat[7] + mat[15] * mat[6] * mat[8] - mat[14] * mat[7] * mat[8]) / len;
	inv[5] = (-mat[0] * mat[11] * mat[14] + mat[0] * mat[10] * mat[15] + mat[11] * mat[12] * mat[2] - mat[10] * mat[12] * mat[3] - mat[15] * mat[2] * mat[8] + mat[14] * mat[3] * mat[8]) / len;
	inv[6] = (mat[15] * mat[2] * mat[4] - mat[14] * mat[3] * mat[4] - mat[0] * mat[15] * mat[6] + mat[12] * mat[3] * mat[6] + mat[0] * mat[14] * mat[7] - mat[12] * mat[2] * mat[7]) / len;
	inv[7] = (-mat[11] * mat[2] * mat[4] + mat[10] * mat[3] * mat[4] + mat[0] * mat[11] * mat[6] - mat[0] * mat[10] * mat[7] - mat[3] * mat[6] * mat[8] + mat[2] * mat[7] * mat[8]) / len;
	inv[8] = (-mat[11] * mat[13] * mat[4] + mat[11] * mat[12] * mat[5] - mat[15] * mat[5] * mat[8] + mat[13] * mat[7] * mat[8] + mat[15] * mat[4] * mat[9] - mat[12] * mat[7] * mat[9]) / len;
	inv[9] = (-mat[1] * mat[11] * mat[12] + mat[0] * mat[11] * mat[13] + mat[1] * mat[15] * mat[8] - mat[13] * mat[3] * mat[8] - mat[0] * mat[15] * mat[9] + mat[12] * mat[3] * mat[9]) / len;
	inv[10] = (-mat[1] * mat[15] * mat[4] + mat[13] * mat[3] * mat[4] + mat[0] * mat[15] * mat[5] - mat[12] * mat[3] * mat[5] + mat[1] * mat[12] * mat[7] - mat[0] * mat[13] * mat[7]) / len;
	inv[11] = (mat[1] * mat[11] * mat[4] - mat[0] * mat[11] * mat[5] + mat[3] * mat[5] * mat[8] - mat[1] * mat[7] * mat[8] - mat[3] * mat[4] * mat[9] + mat[0] * mat[7] * mat[9]) / len;
	inv[12] = (mat[10] * mat[13] * mat[4] - mat[10] * mat[12] * mat[5] + mat[14] * mat[5] * mat[8] - mat[13] * mat[6] * mat[8] - mat[14] * mat[4] * mat[9] + mat[12] * mat[6] * mat[9]) / len;
	inv[13] = (mat[1] * mat[10] * mat[12] - mat[0] * mat[10] * mat[13] - mat[1] * mat[14] * mat[8] + mat[13] * mat[2] * mat[8] + mat[0] * mat[14] * mat[9] - mat[12] * mat[2] * mat[9]) / len;
	inv[14] = (mat[1] * mat[14] * mat[4] - mat[13] * mat[2] * mat[4] - mat[0] * mat[14] * mat[5] + mat[12] * mat[2] * mat[5] - mat[1] * mat[12] * mat[6] + mat[0] * mat[13] * mat[6]) / len;
	inv[15] = (-mat[1] * mat[10] * mat[4] + mat[0] * mat[10] * mat[5] - mat[2] * mat[5] * mat[8] + mat[1] * mat[6] * mat[8] + mat[2] * mat[4] * mat[9] - mat[0] * mat[6] * mat[9]) / len;
}

__device__ __forceinline__
void getMutilVVT(float* v, float* mat, int dim)
{
	for(uint32_t i = 0; i < 3; i++)
		for (uint32_t j = 0; j < 3; j++)
			mat[i * 3 + j] = v[i] * v[j];
}

__device__ __forceinline__
void getMutilMV(float*mat, float* v, float* result, int rdim, int cdim)
{
	for (uint32_t r = 0; r < rdim; r++)
	{
		result[r] = 0;
		for (uint32_t c = 0; c < cdim; c++)
			result[r] += mat[c * rdim + r] * v[c];
	}
}

__device__ __forceinline__
void getMatrix3EigenValue(float* mat, float* result)
{
	float u[3];
	u[0] = 1.0;
	u[1] = 1.0;
	u[2] = 1.0;

	float v[3];
	getMutilMV(mat, u, v);
	float mk = fmaxf(fmaxf(CUDA_ABS(v[0]), CUDA_ABS(v[1])), CUDA_ABS(v[2]));
	float mk_ = 1.0;
	int k = 0;
	while (CUDA_ABS(mk - mk_) > FLT_MIN && k < 20)
	{
		u[0] = v[0] / mk;
		u[1] = v[1] / mk;
		u[2] = v[2] / mk;
		getMutilMV(mat, u, v);
		mk_ = mk;
		mk = fmaxf(fmaxf(CUDA_ABS(v[0]), CUDA_ABS(v[1])), CUDA_ABS(v[2]));
		u[0] = v[0] / mk;
		u[1] = v[1] / mk;
		u[2] = v[2] / mk;
		k++;
	}
	*result = mk;
}
